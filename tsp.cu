#include "hip/hip_runtime.h"
/*
TSP code for CS 4380 / CS 5351

Copyright (c) 2016, Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is not permitted. Use in source and binary forms, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" 
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE 
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE 
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE 
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR 
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF 
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN 
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) 
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF 
THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <cmath>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "cs43805351.h"

#define dist(a, b) (int)(sqrtf((px[a] - px[b]) * (px[a] - px[b]) + (py[a] \
                                - py[b]) * (py[a] - py[b])) + 0.5f)
#define ull unsigned long long int
static const int ThreadsPerBlock = 512;

static __global__
void BestMove(const int cities, float px[], float py[], long* minchange)
{
    const int idx = threadIdx.x + blockIdx.x * blockDim.x;
    const int i = idx / cities;
    const int j = idx % cities;
    if ((i < cities - 2) && (j >= i + 2)) {
        long change = dist(i, j) + dist(i + 1, j + 1) \
                    - dist(i, i + 1) - dist(j, j + 1);
        change = (change << 32) + (i << 16) + j;
        long tmp;
        do {
            tmp = *minchange;
        } while((change < tmp) && \
                (atomicCAS((ull*)minchange, (ull)tmp, (ull)change) \
                    != (ull)tmp));
    }
}

static __global__
void ApplyMove(const long minchange, float px[], float py[])
{
    //todo: apply the best move using just a single thread
    int i = (minchange >> 16) & 0xffff;
    int j = minchange & 0xffff;
    ++i;
    while (i < j) {
        float t;
        t = px[i];
        px[i] = px[j];
        px[j] = t;
        t = py[i];
        py[i] = py[j];
        py[j] = t;
        ++i;
        --j;
    }
}

static int TwoOpt(int cities, float px[], float py[], int& climbs)
{
    // link end to beginning
    px[cities] = px[0];
    py[cities] = py[0];

    float* px_d;
    float* py_d;
    long* minchange_d; // GPU's counterpart of minchange
    if (hipSuccess != \
          hipMalloc((void **)&px_d, (cities + 1) * sizeof(float))) {
        fprintf(stderr, "could not allocate px\n"); 
        exit(-1);
    }

    // allocate py array on GPU
    if (hipSuccess != \
          hipMalloc((void **)&py_d, (cities + 1) * sizeof(float))) {
        fprintf(stderr, "could not allocate py\n");
        exit(-1);
    } 
    if (hipSuccess != \
          hipMalloc((void **)&minchange_d, sizeof(long))) {
        fprintf(stderr, "could not allocate minchange\n"); 
        exit(-1);
    }

    // copy the px and py arrays to the GPU
    if (hipSuccess != \ // FIXME &px -> py
          hipMemcpy(px_d, px, (cities + 1) * sizeof(float), hipMemcpyHostToDevice)) {
        fprintf(stderr, "px copying to device failed\n");
        exit(-1);
    }
    if (hipSuccess != \
          hipMemcpy(py_d, py, (cities + 1) * sizeof(float), hipMemcpyHostToDevice)) {
        fprintf(stderr, "py copying to device failed\n");
        exit(-1);
    }

    // repeat until no improvement
    long minchange;
    int iter = 0;
    do {
        iter++;

        // determine best 2-opt move
        minchange = 0;
        if (hipSuccess != \
              hipMemcpy(minchange_d, &minchange, sizeof(long), \
                hipMemcpyHostToDevice)) {
            fprintf(stderr, "minchange copying to device failed\n"); 
            exit(-1);
        }
        BestMove<<<(cities * cities + ThreadsPerBlock - 1) \
          / ThreadsPerBlock, ThreadsPerBlock>>>(cities, px_d, py_d, \
            minchange_d);
        // copy minchange back to the CPU
        if (hipSuccess != \
              hipMemcpy(&minchange, minchange_d, sizeof(long), \
		hipMemcpyDeviceToHost)) {
            fprintf(stderr, "minchange copying to host failed\n");
            exit(-1);
        }

        // apply move if it shortens the tour
        if (minchange < 0) {
            // calling the ApplyMove kernel here
            ApplyMove<<<1,1>>>(minchange, px_d, py_d);
        }
    } while (minchange < 0);
    climbs = iter;

    //copying the px and py arrays back to the CPU
    if (hipSuccess != \
          hipMemcpy(px, px_d, (cities + 1) * sizeof(float), \
	    hipMemcpyDeviceToHost)) {
        fprintf(stderr, "px copying to host failed\n");
        exit(-1);
    }
    if (hipSuccess != \
          hipMemcpy(py, py_d, (cities + 1) * sizeof(float), \
	    hipMemcpyDeviceToHost)) {
        fprintf(stderr, "py copying to host failed\n");
    }

    //free all dynamically allocated GPU memory
    hipFree(px_d);
    hipFree(py_d);
    hipFree(minchange_d);

    // compute tour length
    int len = 0;
    for (int i = 0; i < cities; i++) {
        len += dist(i, i + 1);
    }
    return len;
}

int main(int argc, char *argv[])
{
    printf("TSP v1.4 [CUDA]\n");

    // read input
    if (argc != 2) {
        fprintf(stderr, "usage: %s input_file\n", argv[0]); 
        exit(-1);
    }
    FILE* f = fopen(argv[1], "rb");  
    if (f == NULL) {
        fprintf(stderr, "error: could not open file %s\n", argv[1]); 
        exit(-1);
    }
    int cities;
    int cnt = fread(&cities, sizeof(int), 1, f);  
    if (cnt != 1) {
        fprintf(stderr, "error: failed to read cities\n"); 
        exit(-1);
    }
    if (cities < 1) {
        fprintf(stderr, "error: cities must be greater than zero\n"); 
        exit(-1);
    }
    float posx[cities + 1], 
          posy[cities + 1];  // need an extra element later
    cnt = fread(posx, sizeof(float), cities, f);  
    if (cnt != cities) {
        fprintf(stderr, "error: failed to read posx\n"); 
        exit(-1);
    }
    cnt = fread(posy, sizeof(float), cities, f);  
    if (cnt != cities) {
        fprintf(stderr, "error: failed to read posy\n"); 
        exit(-1);
    }
    fclose(f);
    printf("configuration: %d cities from %s\n", cities, argv[1]);

    // start time
    struct timeval start, end;
    gettimeofday(&start, NULL);

    // find good tour
    int climbs;
    int len = TwoOpt(cities, posx, posy, climbs);

    // end time
    gettimeofday(&end, NULL);
    double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec \
                   - start.tv_usec / 1000000.0;
    printf("compute time: %.4f s\n", runtime);
    long moves = 1LL * climbs * (cities - 2) * (cities - 1) / 2;
    printf("gigamoves/sec: %.3f\n", moves * 0.000000001 / runtime);

    // output result
    printf("tour length = %d\n", len);

    // scale and draw final tour
    const int width = 1024;
    unsigned char pic[width][width];
    memset(pic, 0, width * width * sizeof(unsigned char));
    float minx = posx[0], 
          maxx = posx[0];
    float miny = posy[0], 
          maxy = posy[0];
    for (int i = 1; i < cities; i++) {
        if (minx > posx[i]) {
             minx = posx[i];
        }
        if (maxx < posx[i]) {
            maxx = posx[i];
        }
        if (miny > posy[i]) { 
            miny = posy[i];
        }
        if (maxy < posy[i]) {
            maxy = posy[i];
        }
    }
    float dist = maxx - minx;
    if (dist < (maxy - miny)) {
        dist = maxy - miny;
    }
    float factor = (width - 1) / dist;
    int x[cities], 
        y[cities];
    for (int i = 0; i < cities; i++) {
        x[i] = (int)(0.5f + (posx[i] - minx) * factor);
        y[i] = (int)(0.5f + (posy[i] - miny) * factor);
    }
    for (int i = 1; i < cities; i++) {
        line(x[i - 1], y[i - 1], x[i], y[i], 127, \
             (unsigned char*)pic, width);
    }
    line(x[cities - 1], y[cities - 1], x[0], y[0], 128, \
         (unsigned char*)pic, width);
    for (int i = 0; i < cities; i++) {
        line(x[i], y[i], x[i], y[i], 255, (unsigned char*)pic, width);
    }
    writeBMP(width, width, (unsigned char*)pic, "tsp.bmp");

    return 0;
}
